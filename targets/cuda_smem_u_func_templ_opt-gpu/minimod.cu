#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"
#include "../../grid.h"

#define N_RADIUS 4
#define N_THREADS_X_DIM 32
#define N_THREADS_Y_DIM 4
#define N_THREADS_Z_DIM 4

// Constant memory coefficients
__constant__ float c_coef0;
__constant__ float c_coefx[N_RADIUS+1];
__constant__ float c_coefy[N_RADIUS+1];
__constant__ float c_coefz[N_RADIUS+1];

#if ENABLE_MEMCPY_ASYNC
#include <cuda_pipeline.h>
#endif

template <int N_T_X, int N_T_Y, int N_T_Z>
__global__ void kernel_7r_3d_inner(
    llint nx, llint ny, llint nz, int ldimx, int ldimy, int ldimz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_Z_DIM+2*N_RADIUS][N_THREADS_Y_DIM+2*N_RADIUS][N_THREADS_X_DIM+2*N_RADIUS];

    const llint i = x3 + blockIdx.z * blockDim.z + threadIdx.z;
    const llint j = y3 + blockIdx.y * blockDim.y + threadIdx.y;
    const llint k = z3 + blockIdx.x * blockDim.x + threadIdx.x;

    const llint sui = threadIdx.z + N_RADIUS;
    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    #if ENABLE_MEMCPY_ASYNC

    __pipeline_memcpy_async(&s_u[sui][suj][suk], &u[IDX3(i,j,k)], sizeof(float));
    if (N_T_X <= N_RADIUS || threadIdx.x < N_RADIUS)
    {
        __pipeline_memcpy_async(&s_u[sui][suj][suk-N_RADIUS], &u[IDX3(i,j,k-N_RADIUS)], sizeof(float));
        __pipeline_memcpy_async(&s_u[sui][suj][suk+N_THREADS_X_DIM], &u[IDX3(i,j,k+N_THREADS_X_DIM)], sizeof(float));
    }
    if (N_T_Y <= N_RADIUS || threadIdx.y < N_RADIUS)
    {
        __pipeline_memcpy_async(&s_u[sui][suj-N_RADIUS][suk], &u[IDX3(i,j-N_RADIUS,k)], sizeof(float));
        __pipeline_memcpy_async(&s_u[sui][suj+N_THREADS_Y_DIM][suk], &u[IDX3(i,j+N_THREADS_Y_DIM,k)], sizeof(float));
    }
    if (N_T_Z <= N_RADIUS || threadIdx.z < N_RADIUS)
    {
        __pipeline_memcpy_async(&s_u[sui-N_RADIUS][suj][suk], &u[IDX3(i-N_RADIUS,j,k)], sizeof(float));
        __pipeline_memcpy_async(&s_u[sui+N_THREADS_Z_DIM][suj][suk], &u[IDX3(i+N_THREADS_Z_DIM,j,k)], sizeof(float));
    }
    __pipeline_commit();
    __pipeline_wait_prior(0);

    __syncthreads();

    #else

    s_u[sui][suj][suk] = u[IDX3(i,j,k)];
    if (N_T_X <= N_RADIUS || threadIdx.x < N_RADIUS)
    {
        s_u[sui][suj][suk-N_RADIUS] = u[IDX3(i,j,k-N_RADIUS)];
        s_u[sui][suj][suk+N_THREADS_X_DIM] = u[IDX3(i,j,k+N_THREADS_X_DIM)];
    }
    if (N_T_Y <= N_RADIUS || threadIdx.y < N_RADIUS)
    {
        s_u[sui][suj-N_RADIUS][suk] = u[IDX3(i,j-N_RADIUS,k)];
        s_u[sui][suj+N_THREADS_Y_DIM][suk] = u[IDX3(i,j+N_THREADS_Y_DIM,k)];
    }
    if (N_T_Z <= N_RADIUS || threadIdx.z < N_RADIUS)
    {
        s_u[sui-N_RADIUS][suj][suk] = u[IDX3(i-N_RADIUS,j,k)];
        s_u[sui+N_THREADS_Z_DIM][suj][suk] = u[IDX3(i+N_THREADS_Z_DIM,j,k)];
    }

    __syncthreads();

    #endif

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = __fmaf_rn(c_coef0, s_u[sui][suj][suk]
              , __fmaf_rn(c_coefx[1], __fadd_rn(s_u[sui+1][suj][suk],s_u[sui-1][suj][suk])
              , __fmaf_rn(c_coefy[1], __fadd_rn(s_u[sui][suj+1][suk],s_u[sui][suj-1][suk])
              , __fmaf_rn(c_coefz[1], __fadd_rn(s_u[sui][suj][suk+1],s_u[sui][suj][suk-1])
              , __fmaf_rn(c_coefx[2], __fadd_rn(s_u[sui+2][suj][suk],s_u[sui-2][suj][suk])
              , __fmaf_rn(c_coefy[2], __fadd_rn(s_u[sui][suj+2][suk],s_u[sui][suj-2][suk])
              , __fmaf_rn(c_coefz[2], __fadd_rn(s_u[sui][suj][suk+2],s_u[sui][suj][suk-2])
              , __fmaf_rn(c_coefx[3], __fadd_rn(s_u[sui+3][suj][suk],s_u[sui-3][suj][suk])
              , __fmaf_rn(c_coefy[3], __fadd_rn(s_u[sui][suj+3][suk],s_u[sui][suj-3][suk])
              , __fmaf_rn(c_coefz[3], __fadd_rn(s_u[sui][suj][suk+3],s_u[sui][suj][suk-3])
              , __fmaf_rn(c_coefx[4], __fadd_rn(s_u[sui+4][suj][suk],s_u[sui-4][suj][suk])
              , __fmaf_rn(c_coefy[4], __fadd_rn(s_u[sui][suj+4][suk],s_u[sui][suj-4][suk])
              , __fmul_rn(c_coefz[4], __fadd_rn(s_u[sui][suj][suk+4],s_u[sui][suj][suk-4])
    )))))))))))));

    v[IDX3(i,j,k)] = __fmaf_rn(2.f, s_u[sui][suj][suk],
        __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3(i,j,k)])
    );
}

template <int N_T_X, int N_T_Y, int N_T_Z>
__global__ void kernel_7r_3d_pml(
    llint nx, llint ny, llint nz, int ldimx, int ldimy, int ldimz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_Z_DIM+2*N_RADIUS][N_THREADS_Y_DIM+2*N_RADIUS][N_THREADS_X_DIM+2*N_RADIUS];

    const llint i = x3 + blockIdx.z * blockDim.z + threadIdx.z;
    const llint j = y3 + blockIdx.y * blockDim.y + threadIdx.y;
    const llint k = z3 + blockIdx.x * blockDim.x + threadIdx.x;

    const llint sui = threadIdx.z + N_RADIUS;
    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    #if ENABLE_MEMCPY_ASYNC

    __pipeline_memcpy_async(&s_u[sui][suj][suk], &u[IDX3(i,j,k)], sizeof(float));
    if (N_T_X <= N_RADIUS || threadIdx.x < N_RADIUS)
    {
        __pipeline_memcpy_async(&s_u[sui][suj][suk-N_RADIUS], &u[IDX3(i,j,k-N_RADIUS)], sizeof(float));
        __pipeline_memcpy_async(&s_u[sui][suj][suk+N_THREADS_X_DIM], &u[IDX3(i,j,k+N_THREADS_X_DIM)], sizeof(float));
    }
    if (N_T_Y <= N_RADIUS || threadIdx.y < N_RADIUS)
    {
        __pipeline_memcpy_async(&s_u[sui][suj-N_RADIUS][suk], &u[IDX3(i,j-N_RADIUS,k)], sizeof(float));
        __pipeline_memcpy_async(&s_u[sui][suj+N_THREADS_Y_DIM][suk], &u[IDX3(i,j+N_THREADS_Y_DIM,k)], sizeof(float));
    }
    if (N_T_Z <= N_RADIUS || threadIdx.z < N_RADIUS)
    {
        __pipeline_memcpy_async(&s_u[sui-N_RADIUS][suj][suk], &u[IDX3(i-N_RADIUS,j,k)], sizeof(float));
        __pipeline_memcpy_async(&s_u[sui+N_THREADS_Z_DIM][suj][suk], &u[IDX3(i+N_THREADS_Z_DIM,j,k)], sizeof(float));
    }
    __pipeline_commit();
    __pipeline_wait_prior(0);

    __syncthreads();

    #else

    s_u[sui][suj][suk] = u[IDX3(i,j,k)];
    if (N_T_X <= N_RADIUS || threadIdx.x < N_RADIUS)
    {
        s_u[sui][suj][suk-N_RADIUS] = u[IDX3(i,j,k-N_RADIUS)];
        s_u[sui][suj][suk+N_THREADS_X_DIM] = u[IDX3(i,j,k+N_THREADS_X_DIM)];
    }
    if (N_T_Y <= N_RADIUS || threadIdx.y < N_RADIUS)
    {
        s_u[sui][suj-N_RADIUS][suk] = u[IDX3(i,j-N_RADIUS,k)];
        s_u[sui][suj+N_THREADS_Y_DIM][suk] = u[IDX3(i,j+N_THREADS_Y_DIM,k)];
    }
    if (N_T_Z <= N_RADIUS || threadIdx.z < N_RADIUS)
    {
        s_u[sui-N_RADIUS][suj][suk] = u[IDX3(i-N_RADIUS,j,k)];
        s_u[sui+N_THREADS_Z_DIM][suj][suk] = u[IDX3(i+N_THREADS_Z_DIM,j,k)];
    }

    __syncthreads();

    #endif

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = __fmaf_rn(c_coef0, s_u[sui][suj][suk]
              , __fmaf_rn(c_coefx[1], __fadd_rn(s_u[sui+1][suj][suk],s_u[sui-1][suj][suk])
              , __fmaf_rn(c_coefy[1], __fadd_rn(s_u[sui][suj+1][suk],s_u[sui][suj-1][suk])
              , __fmaf_rn(c_coefz[1], __fadd_rn(s_u[sui][suj][suk+1],s_u[sui][suj][suk-1])
              , __fmaf_rn(c_coefx[2], __fadd_rn(s_u[sui+2][suj][suk],s_u[sui-2][suj][suk])
              , __fmaf_rn(c_coefy[2], __fadd_rn(s_u[sui][suj+2][suk],s_u[sui][suj-2][suk])
              , __fmaf_rn(c_coefz[2], __fadd_rn(s_u[sui][suj][suk+2],s_u[sui][suj][suk-2])
              , __fmaf_rn(c_coefx[3], __fadd_rn(s_u[sui+3][suj][suk],s_u[sui-3][suj][suk])
              , __fmaf_rn(c_coefy[3], __fadd_rn(s_u[sui][suj+3][suk],s_u[sui][suj-3][suk])
              , __fmaf_rn(c_coefz[3], __fadd_rn(s_u[sui][suj][suk+3],s_u[sui][suj][suk-3])
              , __fmaf_rn(c_coefx[4], __fadd_rn(s_u[sui+4][suj][suk],s_u[sui-4][suj][suk])
              , __fmaf_rn(c_coefy[4], __fadd_rn(s_u[sui][suj+4][suk],s_u[sui][suj-4][suk])
              , __fmul_rn(c_coefz[4], __fadd_rn(s_u[sui][suj][suk+4],s_u[sui][suj][suk-4])
    )))))))))))));

    const float s_eta_c = eta[IDX3(i,j,k)];

    v[IDX3(i,j,k)] = __fdiv_rn(
        __fmaf_rn(
            __fmaf_rn(2.f, s_eta_c,
                __fsub_rn(2.f,
                    __fmul_rn(s_eta_c, s_eta_c)
                )
            ),
            s_u[sui][suj][suk],
            __fmaf_rn(
                vp[IDX3(i,j,k)],
                __fadd_rn(lap, phi[IDX3(i,j,k)]),
                -v[IDX3(i,j,k)]
            )
        ),
        __fmaf_rn(2.f, s_eta_c, 1.f)
    );

    phi[IDX3(i,j,k)] = __fdiv_rn(
            __fsub_rn(
                phi[IDX3(i,j,k)],
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(eta[IDX3(i+1,j,k)], eta[IDX3(i-1,j,k)]),
                    __fsub_rn(s_u[sui+1][suj][suk], s_u[sui-1][suj][suk])
                ), hdx_2,
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(eta[IDX3(i,j+1,k)], eta[IDX3(i,j-1,k)]),
                    __fsub_rn(s_u[sui][suj+1][suk], s_u[sui][suj-1][suk])
                ), hdy_2,
                __fmul_rn(
                    __fmul_rn(
                        __fsub_rn(eta[IDX3(i,j,k+1)], eta[IDX3(i,j,k-1)]),
                        __fsub_rn(s_u[sui][suj][suk+1], s_u[sui][suj][suk-1])
                    ),
                hdz_2)
                ))
            )
        ,
        __fadd_rn(1.f, s_eta_c)
    );
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    const grid_t grid,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    // printf("pre-computation: %f %f %f\n", hdx_2, hdy_2, hdz_2);

    struct timespec start, end;

    float *d_u = allocateDeviceGrid(grid);
    float *d_v = allocateDeviceGrid(grid);
    float *d_phi = allocateDeviceGrid(grid);
    float *d_eta = allocateDeviceGrid(grid);
    float *d_vp = allocateDeviceGrid(grid);

    hipMemset (d_u, 0, gridSize(grid));
    hipMemset (d_v, 0, gridSize(grid));
    hipMemcpy(d_vp, vp, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_phi, phi, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_eta, eta, gridSize(grid), hipMemcpyDefault);

    float coef0 = coefx[0] + coefy[0] + coefz[0];
    hipMemcpyToSymbol(HIP_SYMBOL(c_coef0), &coef0, sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefx), coefx, (N_RADIUS + 1) * sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefy), coefy, (N_RADIUS + 1) * sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefz), coefz, (N_RADIUS + 1) * sizeof (float));

    const llint xmin = 0; const llint xmax = grid.nx;
    const llint ymin = 0; const llint ymax = grid.ny;

    dim3 threadsPerBlock(N_THREADS_X_DIM, N_THREADS_Y_DIM, N_THREADS_Z_DIM);

    int num_streams = 1;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&(streams[i]));
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (grid.z2-grid.z1+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.ny+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            (grid.nx+N_THREADS_Z_DIM-1) / N_THREADS_Z_DIM);
        kernel_7r_3d_pml<N_THREADS_X_DIM,N_THREADS_Y_DIM,N_THREADS_Z_DIM><<<n_block_front, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin, xmax, ymin, ymax, grid.z1, grid.z2,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y2-grid.y1+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            (grid.nx+N_THREADS_Z_DIM-1) / N_THREADS_Z_DIM);
        kernel_7r_3d_pml<N_THREADS_X_DIM,N_THREADS_Y_DIM,N_THREADS_Z_DIM><<<n_block_top, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,grid.y1,grid.y2,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y4-grid.y3+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            (grid.x2-grid.x1+N_THREADS_Z_DIM-1) / N_THREADS_Z_DIM);
        kernel_7r_3d_pml<N_THREADS_X_DIM,N_THREADS_Y_DIM,N_THREADS_Z_DIM><<<n_block_left, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x1,grid.x2,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y4-grid.y3+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            (grid.x4-grid.x3+N_THREADS_Z_DIM-1) / N_THREADS_Z_DIM);
        kernel_7r_3d_inner<N_THREADS_X_DIM,N_THREADS_Y_DIM,N_THREADS_Z_DIM><<<n_block_center, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x3,grid.x4,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y4-grid.y3+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            (grid.x6-grid.x5+N_THREADS_Z_DIM-1) / N_THREADS_Z_DIM);
        kernel_7r_3d_pml<N_THREADS_X_DIM,N_THREADS_Y_DIM,N_THREADS_Z_DIM><<<n_block_right, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x5,grid.x6,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y6-grid.y5+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            (grid.nx+N_THREADS_Z_DIM-1) / N_THREADS_Z_DIM);
        kernel_7r_3d_pml<N_THREADS_X_DIM,N_THREADS_Y_DIM,N_THREADS_Z_DIM><<<n_block_bottom, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,grid.y5,grid.y6,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (grid.z6-grid.z5+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.ny+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            (grid.nx+N_THREADS_Z_DIM-1) / N_THREADS_Z_DIM);
        kernel_7r_3d_pml<N_THREADS_X_DIM,N_THREADS_Y_DIM,N_THREADS_Z_DIM><<<n_block_back, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,ymin,ymax,grid.z5,grid.z6,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_grid(sx,sy,sz,grid), source[istep-1]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, gridSize(grid), hipMemcpyDeviceToHost);

    freeDeviceGrid(d_u, grid);
    freeDeviceGrid(d_v, grid);
    freeDeviceGrid(d_vp, grid);
    freeDeviceGrid(d_phi, grid);
    freeDeviceGrid(d_eta, grid);
}
