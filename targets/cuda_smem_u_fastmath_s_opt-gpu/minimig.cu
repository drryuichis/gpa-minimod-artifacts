#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"
#include "../../grid.h"

#define N_RADIUS 4
#define N_THREADS_PER_BLOCK_DIM 8

__global__ void target_inner_3d_kernel(
    llint nx, llint ny, llint nz, int ldimx, int ldimy, int ldimz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS];

    const llint i0 = x3 + blockIdx.z * blockDim.z;
    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint i = i0 + threadIdx.z;
    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sui = threadIdx.z + N_RADIUS;
    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    const int z_side = threadIdx.z / N_RADIUS;
    s_u[threadIdx.z+z_side*N_THREADS_PER_BLOCK_DIM][suj][suk] = u[IDX3(i0+threadIdx.z+(z_side*2-1)*N_RADIUS,j,k)];
    const int y_side = threadIdx.y / N_RADIUS;
    s_u[sui][threadIdx.y+y_side*N_THREADS_PER_BLOCK_DIM][suk] = u[IDX3(i,j0+threadIdx.y+(y_side*2-1)*N_RADIUS,k)];
    s_u[sui][suj][threadIdx.x] = u[IDX3(i,j,k0+threadIdx.x-N_RADIUS)];
    s_u[sui][suj][threadIdx.x+N_THREADS_PER_BLOCK_DIM] = u[IDX3(i,j,k0+threadIdx.x+N_RADIUS)];

    __syncthreads();

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = __fmaf_rn(coef0, s_u[sui][suj][suk]
              , __fmaf_rn(coefx_1, __fadd_rn(s_u[sui+1][suj][suk],s_u[sui-1][suj][suk])
              , __fmaf_rn(coefy_1, __fadd_rn(s_u[sui][suj+1][suk],s_u[sui][suj-1][suk])
              , __fmaf_rn(coefz_1, __fadd_rn(s_u[sui][suj][suk+1],s_u[sui][suj][suk-1])
              , __fmaf_rn(coefx_2, __fadd_rn(s_u[sui+2][suj][suk],s_u[sui-2][suj][suk])
              , __fmaf_rn(coefy_2, __fadd_rn(s_u[sui][suj+2][suk],s_u[sui][suj-2][suk])
              , __fmaf_rn(coefz_2, __fadd_rn(s_u[sui][suj][suk+2],s_u[sui][suj][suk-2])
              , __fmaf_rn(coefx_3, __fadd_rn(s_u[sui+3][suj][suk],s_u[sui-3][suj][suk])
              , __fmaf_rn(coefy_3, __fadd_rn(s_u[sui][suj+3][suk],s_u[sui][suj-3][suk])
              , __fmaf_rn(coefz_3, __fadd_rn(s_u[sui][suj][suk+3],s_u[sui][suj][suk-3])
              , __fmaf_rn(coefx_4, __fadd_rn(s_u[sui+4][suj][suk],s_u[sui-4][suj][suk])
              , __fmaf_rn(coefy_4, __fadd_rn(s_u[sui][suj+4][suk],s_u[sui][suj-4][suk])
              , __fmul_rn(coefz_4, __fadd_rn(s_u[sui][suj][suk+4],s_u[sui][suj][suk-4])
    )))))))))))));

    v[IDX3(i,j,k)] = __fmaf_rn(2.f, s_u[sui][suj][suk],
        __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3(i,j,k)])
    );
}

__global__ void target_pml_3d_kernel(
    llint nx, llint ny, llint nz, int ldimx, int ldimy, int ldimz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    float coef0,
    float coefx_1, float coefx_2, float coefx_3, float coefx_4,
    float coefy_1, float coefy_2, float coefy_3, float coefy_4,
    float coefz_1, float coefz_2, float coefz_3, float coefz_4,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS][N_THREADS_PER_BLOCK_DIM+2*N_RADIUS];

    const llint i0 = x3 + blockIdx.z * blockDim.z;
    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint i = i0 + threadIdx.z;
    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sui = threadIdx.z + N_RADIUS;
    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    const int z_side = threadIdx.z / N_RADIUS;
    s_u[threadIdx.z+z_side*N_THREADS_PER_BLOCK_DIM][suj][suk] = u[IDX3(i0+threadIdx.z+(z_side*2-1)*N_RADIUS,j,k)];
    const int y_side = threadIdx.y / N_RADIUS;
    s_u[sui][threadIdx.y+y_side*N_THREADS_PER_BLOCK_DIM][suk] = u[IDX3(i,j0+threadIdx.y+(y_side*2-1)*N_RADIUS,k)];
    s_u[sui][suj][threadIdx.x] = u[IDX3(i,j,k0+threadIdx.x-N_RADIUS)];
    s_u[sui][suj][threadIdx.x+N_THREADS_PER_BLOCK_DIM] = u[IDX3(i,j,k0+threadIdx.x+N_RADIUS)];

    __syncthreads();

    if (i > x4-1 || j > y4-1 || k > z4-1) { return; }

    float lap = __fmaf_rn(coef0, s_u[sui][suj][suk]
        , __fmaf_rn(coefx_1, __fadd_rn(s_u[sui+1][suj][suk],s_u[sui-1][suj][suk])
        , __fmaf_rn(coefy_1, __fadd_rn(s_u[sui][suj+1][suk],s_u[sui][suj-1][suk])
        , __fmaf_rn(coefz_1, __fadd_rn(s_u[sui][suj][suk+1],s_u[sui][suj][suk-1])
        , __fmaf_rn(coefx_2, __fadd_rn(s_u[sui+2][suj][suk],s_u[sui-2][suj][suk])
        , __fmaf_rn(coefy_2, __fadd_rn(s_u[sui][suj+2][suk],s_u[sui][suj-2][suk])
        , __fmaf_rn(coefz_2, __fadd_rn(s_u[sui][suj][suk+2],s_u[sui][suj][suk-2])
        , __fmaf_rn(coefx_3, __fadd_rn(s_u[sui+3][suj][suk],s_u[sui-3][suj][suk])
        , __fmaf_rn(coefy_3, __fadd_rn(s_u[sui][suj+3][suk],s_u[sui][suj-3][suk])
        , __fmaf_rn(coefz_3, __fadd_rn(s_u[sui][suj][suk+3],s_u[sui][suj][suk-3])
        , __fmaf_rn(coefx_4, __fadd_rn(s_u[sui+4][suj][suk],s_u[sui-4][suj][suk])
        , __fmaf_rn(coefy_4, __fadd_rn(s_u[sui][suj+4][suk],s_u[sui][suj-4][suk])
        , __fmul_rn(coefz_4, __fadd_rn(s_u[sui][suj][suk+4],s_u[sui][suj][suk-4])
    )))))))))))));

    const float s_eta_c = eta[IDX3(i,j,k)];

    v[IDX3(i,j,k)] = //__fdiv_rn(
        __fmaf_rn(
            __fmaf_rn(2.f, s_eta_c,
                __fsub_rn(2.f,
                    __fmul_rn(s_eta_c, s_eta_c)
                )
            ),
            s_u[sui][suj][suk],
            __fmaf_rn(
                vp[IDX3(i,j,k)],
                __fadd_rn(lap, phi[IDX3(i,j,k)]),
                -v[IDX3(i,j,k)]
            )
        ) / //,
        __fmaf_rn(2.f, s_eta_c, 1.f)
    ;//);

    phi[IDX3(i,j,k)] = // __fdiv_rn(
            __fsub_rn(
                phi[IDX3(i,j,k)],
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(eta[IDX3(i+1,j,k)], eta[IDX3(i-1,j,k)]),
                    __fsub_rn(s_u[sui+1][suj][suk], s_u[sui-1][suj][suk])
                ), hdx_2,
                __fmaf_rn(
                __fmul_rn(
                    __fsub_rn(eta[IDX3(i,j+1,k)], eta[IDX3(i,j-1,k)]),
                    __fsub_rn(s_u[sui][suj+1][suk], s_u[sui][suj-1][suk])
                ), hdy_2,
                __fmul_rn(
                    __fmul_rn(
                        __fsub_rn(eta[IDX3(i,j,k+1)], eta[IDX3(i,j,k-1)]),
                        __fsub_rn(s_u[sui][suj][suk+1], s_u[sui][suj][suk-1])
                    ),
                hdz_2)
                ))
            )
        / //,
        __fadd_rn(1.f, s_eta_c)
    ; //);
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    const grid_t grid,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    float *d_u = allocateDeviceGrid(grid);
    float *d_v = allocateDeviceGrid(grid);
    float *d_phi = allocateDeviceGrid(grid);
    float *d_eta = allocateDeviceGrid(grid);
    float *d_vp = allocateDeviceGrid(grid);

    hipMemset (d_u, 0, gridSize(grid));
    hipMemset (d_v, 0, gridSize(grid));
    hipMemcpy(d_vp, vp, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_phi, phi, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_eta, eta, gridSize(grid), hipMemcpyDefault);

    const llint xmin = 0; const llint xmax = grid.nx;
    const llint ymin = 0; const llint ymax = grid.ny;

    dim3 threadsPerBlock(N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM, N_THREADS_PER_BLOCK_DIM);

    int num_streams = 1;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&(streams[i]));
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (grid.z2-grid.z1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_front, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin, xmax, ymin, ymax, grid.z1, grid.z2,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (grid.z4-grid.z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.y2-grid.y1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_top, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,grid.y1,grid.y2,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (grid.z4-grid.z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.y4-grid.y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.x2-grid.x1+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_left, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x1,grid.x2,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (grid.z4-grid.z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.y4-grid.y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.x4-grid.x3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_inner_3d_kernel<<<n_block_center, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x3,grid.x4,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (grid.z4-grid.z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.y4-grid.y3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.x6-grid.x5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_right, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x5,grid.x6,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (grid.z4-grid.z3+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.y6-grid.y5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_bottom, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,grid.y5,grid.y6,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (grid.z6-grid.z5+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.ny+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM,
            (grid.nx+N_THREADS_PER_BLOCK_DIM-1) / N_THREADS_PER_BLOCK_DIM);
        target_pml_3d_kernel<<<n_block_back, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,ymin,ymax,grid.z5,grid.z6,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            coefx[0]+coefy[0]+coefz[0],
            coefx[1], coefx[2], coefx[3], coefx[4],
            coefy[1], coefy[2], coefy[3], coefy[4],
            coefz[1], coefz[2], coefz[3], coefz[4],
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_grid(sx,sy,sz,grid), source[istep-1]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, gridSize(grid), hipMemcpyDeviceToHost);

    freeDeviceGrid(d_u, grid);
    freeDeviceGrid(d_v, grid);
    freeDeviceGrid(d_vp, grid);
    freeDeviceGrid(d_phi, grid);
    freeDeviceGrid(d_eta, grid);
}
