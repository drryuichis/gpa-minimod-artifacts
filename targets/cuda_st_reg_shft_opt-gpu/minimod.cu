#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "../../constants.h"
#include "../../grid.h"

#define N_RADIUS 4
#define N_THREADS_X_DIM 32
#define N_THREADS_Y_DIM 32
#define N_THREADS_Z_DIM 0

// Constant memory coefficients
__constant__ float c_coef0;
__constant__ float c_coefx[N_RADIUS+1];
__constant__ float c_coefy[N_RADIUS+1];
__constant__ float c_coefz[N_RADIUS+1];

#ifdef ENABLE_MEMCPY_ASYNC
#include <cuda_pipeline.h>
#endif

__global__ void kernel_7r_25d_inner(
    llint nx, llint ny, llint nz, int ldimx, int ldimy, int ldimz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[2][N_THREADS_Y_DIM+2*N_RADIUS][N_THREADS_X_DIM+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint je = min(j0 + N_THREADS_Y_DIM, y4);
    const llint ke = min(k0 + N_THREADS_X_DIM, z4);

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sje = (j0+N_THREADS_Y_DIM<y4) ? N_THREADS_Y_DIM : ((y4-y3-1)%N_THREADS_Y_DIM+1);
    const llint ske = (k0+N_THREADS_X_DIM<z4) ? N_THREADS_X_DIM : ((z4-z3-1)%N_THREADS_X_DIM+1);

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    float infront1, infront2, infront3, infront4; // variables for input “in front of” the current slice
    float behind1, behind2, behind3, behind4; // variables for input “behind” the current slice
    float current; // input value in the current slice

    behind3  = u[IDX3(x3-4,j,k)];
    behind2  = u[IDX3(x3-3,j,k)];
    behind1  = u[IDX3(x3-2,j,k)];
    current  = u[IDX3(x3-1,j,k)];
    infront1 = u[IDX3(x3+0,j,k)];
    infront2 = u[IDX3(x3+1,j,k)];
    infront3 = u[IDX3(x3+2,j,k)];
    infront4 = u[IDX3(x3+3,j,k)];

    int double_buffer_current = 0, double_buffer_next = 1;

    #ifdef ENABLE_MEMCPY_ASYNC

    if (threadIdx.y < N_RADIUS) {
        __pipeline_memcpy_async(&s_u[double_buffer_current][threadIdx.y][suk], &u[IDX3(x3, j - N_RADIUS, k)], sizeof(float));
        __pipeline_memcpy_async(&s_u[double_buffer_current][threadIdx.y+sje+N_RADIUS][suk], &u[IDX3(x3, threadIdx.y+je, k)], sizeof(float));
    }
    if (threadIdx.x < N_RADIUS) {
        __pipeline_memcpy_async(&s_u[double_buffer_current][suj][threadIdx.x], &u[IDX3(x3,j,k - N_RADIUS)], sizeof(float));
        __pipeline_memcpy_async(&s_u[double_buffer_current][suj][threadIdx.x+ske+N_RADIUS], &u[IDX3(x3,j,threadIdx.x+ke)], sizeof(float));
    }
    __pipeline_memcpy_async(&s_u[double_buffer_current][suj][suk], &u[IDX3(x3,j,k)], sizeof(float));
    __pipeline_commit();


    for (llint i = x3; i < x4; i++) {
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = u[IDX3(i+N_RADIUS,j,k)];

        __pipeline_wait_prior(0);

        __syncthreads();
        
        if (i+1 < x4) {
            if (threadIdx.y < N_RADIUS) {
                __pipeline_memcpy_async(&s_u[double_buffer_next][threadIdx.y][suk], &u[IDX3(i+1, j - N_RADIUS, k)], sizeof(float));
                __pipeline_memcpy_async(&s_u[double_buffer_next][threadIdx.y+sje+N_RADIUS][suk], &u[IDX3(i+1, threadIdx.y+je, k)], sizeof(float));
            }
            if (threadIdx.x < N_RADIUS) {
                __pipeline_memcpy_async(&s_u[double_buffer_next][suj][threadIdx.x], &u[IDX3(i+1,j,k - N_RADIUS)], sizeof(float));
                __pipeline_memcpy_async(&s_u[double_buffer_next][suj][threadIdx.x+ske+N_RADIUS], &u[IDX3(i+1,j,threadIdx.x+ke)], sizeof(float));
            }
            __pipeline_memcpy_async(&s_u[double_buffer_next][suj][suk], &u[IDX3(i+1,j,k)], sizeof(float));
            __pipeline_commit();
        }

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(c_coef0, current
                      , __fmaf_rn(c_coefx[1], __fadd_rn(infront1,behind1)
                      , __fmaf_rn(c_coefy[1], __fadd_rn(s_u[double_buffer_current][suj+1][suk],s_u[double_buffer_current][suj-1][suk])
                      , __fmaf_rn(c_coefz[1], __fadd_rn(s_u[double_buffer_current][suj][suk+1],s_u[double_buffer_current][suj][suk-1])
                      , __fmaf_rn(c_coefx[2], __fadd_rn(infront2,behind2)
                      , __fmaf_rn(c_coefy[2], __fadd_rn(s_u[double_buffer_current][suj+2][suk],s_u[double_buffer_current][suj-2][suk])
                      , __fmaf_rn(c_coefz[2], __fadd_rn(s_u[double_buffer_current][suj][suk+2],s_u[double_buffer_current][suj][suk-2])
                      , __fmaf_rn(c_coefx[3], __fadd_rn(infront3,behind3)
                      , __fmaf_rn(c_coefy[3], __fadd_rn(s_u[double_buffer_current][suj+3][suk],s_u[double_buffer_current][suj-3][suk])
                      , __fmaf_rn(c_coefz[3], __fadd_rn(s_u[double_buffer_current][suj][suk+3],s_u[double_buffer_current][suj][suk-3])
                      , __fmaf_rn(c_coefx[4], __fadd_rn(infront4,behind4)
                      , __fmaf_rn(c_coefy[4], __fadd_rn(s_u[double_buffer_current][suj+4][suk],s_u[double_buffer_current][suj-4][suk])
                      , __fmul_rn(c_coefz[4], __fadd_rn(s_u[double_buffer_current][suj][suk+4],s_u[double_buffer_current][suj][suk-4])
            )))))))))))));

            v[IDX3(i,j,k)] = __fmaf_rn(2.f, current,
                __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3(i,j,k)])
            );
        }

        double_buffer_current = 1 - double_buffer_current;
        double_buffer_next = 1 - double_buffer_next;
    }

    #else

    if (threadIdx.y < N_RADIUS) {
      s_u[double_buffer_current][threadIdx.y][suk] = u[IDX3(x3, j - N_RADIUS, k)];
      s_u[double_buffer_current][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3(x3, threadIdx.y+je, k)];
    }
    if (threadIdx.x < N_RADIUS) {
      s_u[double_buffer_current][suj][threadIdx.x] = u[IDX3(x3,j,k - N_RADIUS)];
      s_u[double_buffer_current][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3(x3,j,threadIdx.x+ke)];
    }
    s_u[double_buffer_current][suj][suk] = u[IDX3(x3,j,k)];

    for (llint i = x3; i < x4; i++) {
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = u[IDX3(i+N_RADIUS,j,k)];

        __syncthreads();

        if (i+1 < x4) {
          if (threadIdx.y < N_RADIUS) {
              s_u[double_buffer_next][threadIdx.y][suk] = u[IDX3(i+1, j - N_RADIUS, k)];
              s_u[double_buffer_next][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3(i+1, threadIdx.y+je, k)];
          }
          if (threadIdx.x < N_RADIUS) {
              s_u[double_buffer_next][suj][threadIdx.x] = u[IDX3(i+1,j,k - N_RADIUS)];
              s_u[double_buffer_next][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3(i+1,j,threadIdx.x+ke)];
          }
          s_u[double_buffer_next][suj][suk] = u[IDX3(i+1,j,k)];
        }

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(c_coef0, current
                      , __fmaf_rn(c_coefx[1], __fadd_rn(infront1,behind1)
                      , __fmaf_rn(c_coefy[1], __fadd_rn(s_u[double_buffer_current][suj+1][suk],s_u[double_buffer_current][suj-1][suk])
                      , __fmaf_rn(c_coefz[1], __fadd_rn(s_u[double_buffer_current][suj][suk+1],s_u[double_buffer_current][suj][suk-1])
                      , __fmaf_rn(c_coefx[2], __fadd_rn(infront2,behind2)
                      , __fmaf_rn(c_coefy[2], __fadd_rn(s_u[double_buffer_current][suj+2][suk],s_u[double_buffer_current][suj-2][suk])
                      , __fmaf_rn(c_coefz[2], __fadd_rn(s_u[double_buffer_current][suj][suk+2],s_u[double_buffer_current][suj][suk-2])
                      , __fmaf_rn(c_coefx[3], __fadd_rn(infront3,behind3)
                      , __fmaf_rn(c_coefy[3], __fadd_rn(s_u[double_buffer_current][suj+3][suk],s_u[double_buffer_current][suj-3][suk])
                      , __fmaf_rn(c_coefz[3], __fadd_rn(s_u[double_buffer_current][suj][suk+3],s_u[double_buffer_current][suj][suk-3])
                      , __fmaf_rn(c_coefx[4], __fadd_rn(infront4,behind4)
                      , __fmaf_rn(c_coefy[4], __fadd_rn(s_u[double_buffer_current][suj+4][suk],s_u[double_buffer_current][suj-4][suk])
                      , __fmul_rn(c_coefz[4], __fadd_rn(s_u[double_buffer_current][suj][suk+4],s_u[double_buffer_current][suj][suk-4])
            )))))))))))));

            v[IDX3(i,j,k)] = __fmaf_rn(2.f, current,
                __fmaf_rn(vp[IDX3(i,j,k)], lap, -v[IDX3(i,j,k)])
            );
        }

        double_buffer_current = 1 - double_buffer_current;
        double_buffer_next = 1 - double_buffer_next;
    }

    #endif
}

__global__ void kernel_7r_25d_pml(
    llint nx, llint ny, llint nz, int ldimx, int ldimy, int ldimz,
    llint x3, llint x4, llint y3, llint y4, llint z3, llint z4,
    llint lx, llint ly, llint lz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ u, float *__restrict__ v, const float *__restrict__ vp,
    float *__restrict__ phi, const float *__restrict__ eta
) {
    __shared__ float s_u[2][N_THREADS_Y_DIM+2*N_RADIUS][N_THREADS_X_DIM+2*N_RADIUS];

    const llint j0 = y3 + blockIdx.y * blockDim.y;
    const llint k0 = z3 + blockIdx.x * blockDim.x;

    const llint je = min(j0 + N_THREADS_Y_DIM, y4);
    const llint ke = min(k0 + N_THREADS_X_DIM, z4);

    const llint j = j0 + threadIdx.y;
    const llint k = k0 + threadIdx.x;

    const llint sje = (j0+N_THREADS_Y_DIM<y4) ? N_THREADS_Y_DIM : ((y4-y3-1)%N_THREADS_Y_DIM+1);
    const llint ske = (k0+N_THREADS_X_DIM<z4) ? N_THREADS_X_DIM : ((z4-z3-1)%N_THREADS_X_DIM+1);

    const llint suj = threadIdx.y + N_RADIUS;
    const llint suk = threadIdx.x + N_RADIUS;

    float infront1, infront2, infront3, infront4; // variables for input “in front of” the current slice
    float behind1, behind2, behind3, behind4; // variables for input “behind” the current slice
    float current; // input value in the current slice

    behind3  = u[IDX3(x3-4,j,k)];
    behind2  = u[IDX3(x3-3,j,k)];
    behind1  = u[IDX3(x3-2,j,k)];
    current  = u[IDX3(x3-1,j,k)];
    infront1 = u[IDX3(x3+0,j,k)];
    infront2 = u[IDX3(x3+1,j,k)];
    infront3 = u[IDX3(x3+2,j,k)];
    infront4 = u[IDX3(x3+3,j,k)];

    int double_buffer_current = 0, double_buffer_next = 1;

    #ifdef ENABLE_MEMCPY_ASYNC

    if (threadIdx.y < N_RADIUS) {
        __pipeline_memcpy_async(&s_u[double_buffer_current][threadIdx.y][suk], &u[IDX3(x3, j - N_RADIUS, k)], sizeof(float));
        __pipeline_memcpy_async(&s_u[double_buffer_current][threadIdx.y+sje+N_RADIUS][suk], &u[IDX3(x3, threadIdx.y+je, k)], sizeof(float));
    }
    if (threadIdx.x < N_RADIUS) {
        __pipeline_memcpy_async(&s_u[double_buffer_current][suj][threadIdx.x], &u[IDX3(x3,j,k - N_RADIUS)], sizeof(float));
        __pipeline_memcpy_async(&s_u[double_buffer_current][suj][threadIdx.x+ske+N_RADIUS], &u[IDX3(x3,j,threadIdx.x+ke)], sizeof(float));
    }
    __pipeline_memcpy_async(&s_u[double_buffer_current][suj][suk], &u[IDX3(x3,j,k)], sizeof(float));
    __pipeline_commit();

    for (llint i = x3; i < x4; i++) {
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = u[IDX3(i+N_RADIUS,j,k)];

        __pipeline_wait_prior(0);

        __syncthreads();

        if (i+1 < x4) {
            if (threadIdx.y < N_RADIUS) {
                __pipeline_memcpy_async(&s_u[double_buffer_next][threadIdx.y][suk], &u[IDX3(i+1, j - N_RADIUS, k)], sizeof(float));
                __pipeline_memcpy_async(&s_u[double_buffer_next][threadIdx.y+sje+N_RADIUS][suk], &u[IDX3(i+1, threadIdx.y+je, k)], sizeof(float));
            }
            if (threadIdx.x < N_RADIUS) {
                __pipeline_memcpy_async(&s_u[double_buffer_next][suj][threadIdx.x], &u[IDX3(i+1,j,k - N_RADIUS)], sizeof(float));
                __pipeline_memcpy_async(&s_u[double_buffer_next][suj][threadIdx.x+ske+N_RADIUS], &u[IDX3(i+1,j,threadIdx.x+ke)], sizeof(float));
            }
            __pipeline_memcpy_async(&s_u[double_buffer_next][suj][suk], &u[IDX3(i+1,j,k)], sizeof(float));
            __pipeline_commit();
        }

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(c_coef0, current
                      , __fmaf_rn(c_coefx[1], __fadd_rn(infront1,behind1)
                      , __fmaf_rn(c_coefy[1], __fadd_rn(s_u[double_buffer_current][suj+1][suk],s_u[double_buffer_current][suj-1][suk])
                      , __fmaf_rn(c_coefz[1], __fadd_rn(s_u[double_buffer_current][suj][suk+1],s_u[double_buffer_current][suj][suk-1])
                      , __fmaf_rn(c_coefx[2], __fadd_rn(infront2,behind2)
                      , __fmaf_rn(c_coefy[2], __fadd_rn(s_u[double_buffer_current][suj+2][suk],s_u[double_buffer_current][suj-2][suk])
                      , __fmaf_rn(c_coefz[2], __fadd_rn(s_u[double_buffer_current][suj][suk+2],s_u[double_buffer_current][suj][suk-2])
                      , __fmaf_rn(c_coefx[3], __fadd_rn(infront3,behind3)
                      , __fmaf_rn(c_coefy[3], __fadd_rn(s_u[double_buffer_current][suj+3][suk],s_u[double_buffer_current][suj-3][suk])
                      , __fmaf_rn(c_coefz[3], __fadd_rn(s_u[double_buffer_current][suj][suk+3],s_u[double_buffer_current][suj][suk-3])
                      , __fmaf_rn(c_coefx[4], __fadd_rn(infront4,behind4)
                      , __fmaf_rn(c_coefy[4], __fadd_rn(s_u[double_buffer_current][suj+4][suk],s_u[double_buffer_current][suj-4][suk])
                      , __fmul_rn(c_coefz[4], __fadd_rn(s_u[double_buffer_current][suj][suk+4],s_u[double_buffer_current][suj][suk-4])
            )))))))))))));

            const float s_eta_c = eta[IDX3(i,j,k)];

            v[IDX3(i,j,k)] = __fdiv_rn(
                __fmaf_rn(
                    __fmaf_rn(2.f, s_eta_c,
                        __fsub_rn(2.f,
                            __fmul_rn(s_eta_c, s_eta_c)
                        )
                    ),
                    current,
                    __fmaf_rn(
                        vp[IDX3(i,j,k)],
                        __fadd_rn(lap, phi[IDX3(i,j,k)]),
                        -v[IDX3(i,j,k)]
                    )
                ),
                __fmaf_rn(2.f, s_eta_c, 1.f)
            );

            phi[IDX3(i,j,k)] = __fdiv_rn(
                    __fsub_rn(
                        phi[IDX3(i,j,k)],
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3(i+1,j,k)], eta[IDX3(i-1,j,k)]),
                            __fsub_rn(infront1,behind1)
                        ), hdx_2,
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3(i,j+1,k)], eta[IDX3(i,j-1,k)]),
                            __fsub_rn(s_u[double_buffer_current][suj+1][suk], s_u[double_buffer_current][suj-1][suk])
                        ), hdy_2,
                        __fmul_rn(
                            __fmul_rn(
                                __fsub_rn(eta[IDX3(i,j,k+1)], eta[IDX3(i,j,k-1)]),
                                __fsub_rn(s_u[double_buffer_current][suj][suk+1], s_u[double_buffer_current][suj][suk-1])
                            ),
                        hdz_2)
                        ))
                    )
                ,
                __fadd_rn(1.f, s_eta_c)
            );
        }

        double_buffer_current = 1 - double_buffer_current;
        double_buffer_next = 1 - double_buffer_next;
    }

    #else

    if (threadIdx.y < N_RADIUS) {
      s_u[double_buffer_current][threadIdx.y][suk] = u[IDX3(x3, j - N_RADIUS, k)];
      s_u[double_buffer_current][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3(x3, threadIdx.y+je, k)];
    }
    if (threadIdx.x < N_RADIUS) {
      s_u[double_buffer_current][suj][threadIdx.x] = u[IDX3(x3,j,k - N_RADIUS)];
      s_u[double_buffer_current][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3(x3,j,threadIdx.x+ke)];
    }

    s_u[double_buffer_current][suj][suk] = u[IDX3(x3,j,k)];

    for (llint i = x3; i < x4; i++) {
        // advance the slice (move the thread-front)
        behind4  = behind3;
        behind3  = behind2;
        behind2  = behind1;
        behind1  = current;
        current  = infront1;
        infront1 = infront2;
        infront2 = infront3;
        infront3 = infront4;
        infront4 = u[IDX3(i+N_RADIUS,j,k)];

        __syncthreads();

        if (i+1 < x4) {
          if (threadIdx.y < N_RADIUS) {
              s_u[double_buffer_next][threadIdx.y][suk] = u[IDX3(i+1, j - N_RADIUS, k)];
              s_u[double_buffer_next][threadIdx.y+sje+N_RADIUS][suk] = u[IDX3(i+1, threadIdx.y+je, k)];
          }
          if (threadIdx.x < N_RADIUS) {
              s_u[double_buffer_next][suj][threadIdx.x] = u[IDX3(i+1,j,k - N_RADIUS)];
              s_u[double_buffer_next][suj][threadIdx.x+ske+N_RADIUS] = u[IDX3(i+1,j,threadIdx.x+ke)];
          }

          s_u[double_buffer_next][suj][suk] = u[IDX3(i+1,j,k)];
        }

        if (j < y4 && k < z4) {
            float lap = __fmaf_rn(c_coef0, current
                      , __fmaf_rn(c_coefx[1], __fadd_rn(infront1,behind1)
                      , __fmaf_rn(c_coefy[1], __fadd_rn(s_u[double_buffer_current][suj+1][suk],s_u[double_buffer_current][suj-1][suk])
                      , __fmaf_rn(c_coefz[1], __fadd_rn(s_u[double_buffer_current][suj][suk+1],s_u[double_buffer_current][suj][suk-1])
                      , __fmaf_rn(c_coefx[2], __fadd_rn(infront2,behind2)
                      , __fmaf_rn(c_coefy[2], __fadd_rn(s_u[double_buffer_current][suj+2][suk],s_u[double_buffer_current][suj-2][suk])
                      , __fmaf_rn(c_coefz[2], __fadd_rn(s_u[double_buffer_current][suj][suk+2],s_u[double_buffer_current][suj][suk-2])
                      , __fmaf_rn(c_coefx[3], __fadd_rn(infront3,behind3)
                      , __fmaf_rn(c_coefy[3], __fadd_rn(s_u[double_buffer_current][suj+3][suk],s_u[double_buffer_current][suj-3][suk])
                      , __fmaf_rn(c_coefz[3], __fadd_rn(s_u[double_buffer_current][suj][suk+3],s_u[double_buffer_current][suj][suk-3])
                      , __fmaf_rn(c_coefx[4], __fadd_rn(infront4,behind4)
                      , __fmaf_rn(c_coefy[4], __fadd_rn(s_u[double_buffer_current][suj+4][suk],s_u[double_buffer_current][suj-4][suk])
                      , __fmul_rn(c_coefz[4], __fadd_rn(s_u[double_buffer_current][suj][suk+4],s_u[double_buffer_current][suj][suk-4])
            )))))))))))));

            const float s_eta_c = eta[IDX3(i,j,k)];

            v[IDX3(i,j,k)] = __fdiv_rn(
                __fmaf_rn(
                    __fmaf_rn(2.f, s_eta_c,
                        __fsub_rn(2.f,
                            __fmul_rn(s_eta_c, s_eta_c)
                        )
                    ),
                    current,
                    __fmaf_rn(
                        vp[IDX3(i,j,k)],
                        __fadd_rn(lap, phi[IDX3(i,j,k)]),
                        -v[IDX3(i,j,k)]
                    )
                ),
                __fmaf_rn(2.f, s_eta_c, 1.f)
            );

            phi[IDX3(i,j,k)] = __fdiv_rn(
                    __fsub_rn(
                        phi[IDX3(i,j,k)],
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3(i+1,j,k)], eta[IDX3(i-1,j,k)]),
                            __fsub_rn(infront1,behind1)
                        ), hdx_2,
                        __fmaf_rn(
                        __fmul_rn(
                            __fsub_rn(eta[IDX3(i,j+1,k)], eta[IDX3(i,j-1,k)]),
                            __fsub_rn(s_u[double_buffer_current][suj+1][suk], s_u[double_buffer_current][suj-1][suk])
                        ), hdy_2,
                        __fmul_rn(
                            __fmul_rn(
                                __fsub_rn(eta[IDX3(i,j,k+1)], eta[IDX3(i,j,k-1)]),
                                __fsub_rn(s_u[double_buffer_current][suj][suk+1], s_u[double_buffer_current][suj][suk-1])
                            ),
                        hdz_2)
                        ))
                    )
                ,
                __fadd_rn(1.f, s_eta_c)
            );
        }

        double_buffer_current = 1 - double_buffer_current;
        double_buffer_next = 1 - double_buffer_next;
    }

    #endif
}

__global__ void kernel_add_source_kernel(float *g_u, llint idx, float source) {
    g_u[idx] += source;
}

extern "C" void target(
    uint nsteps, double *time_kernel,
    const grid_t grid,
    llint sx, llint sy, llint sz,
    float hdx_2, float hdy_2, float hdz_2,
    const float *__restrict__ coefx, const float *__restrict__ coefy, const float *__restrict__ coefz,
    float *__restrict__ u, const float *__restrict__ v, const float *__restrict__ vp,
    const float *__restrict__ phi, const float *__restrict__ eta, const float *__restrict__ source
) {
    struct timespec start, end;

    float *d_u = allocateDeviceGrid(grid);
    float *d_v = allocateDeviceGrid(grid);
    float *d_phi = allocateDeviceGrid(grid);
    float *d_eta = allocateDeviceGrid(grid);
    float *d_vp = allocateDeviceGrid(grid);

    hipMemset (d_u, 0, gridSize(grid));
    hipMemset (d_v, 0, gridSize(grid));
    hipMemcpy(d_vp, vp, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_phi, phi, gridSize(grid), hipMemcpyDefault);
    hipMemcpy(d_eta, eta, gridSize(grid), hipMemcpyDefault);

    float coef0 = coefx[0] + coefy[0] + coefz[0];
    hipMemcpyToSymbol(HIP_SYMBOL(c_coef0), &coef0, sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefx), coefx, (N_RADIUS + 1) * sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefy), coefy, (N_RADIUS + 1) * sizeof (float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_coefz), coefz, (N_RADIUS + 1) * sizeof (float));

    const llint xmin = 0; const llint xmax = grid.nx;
    const llint ymin = 0; const llint ymax = grid.ny;

    dim3 threadsPerBlock(N_THREADS_X_DIM, N_THREADS_Y_DIM, 1);

    int num_streams = 1;
    hipStream_t streams[num_streams];
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreateWithFlags(&(streams[i]), hipStreamNonBlocking);
    }

    const uint npo = 100;
    for (uint istep = 1; istep <= nsteps; ++istep) {
        clock_gettime(CLOCK_REALTIME, &start);

        dim3 n_block_front(
            (grid.z2-grid.z1+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.ny+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            1);
        kernel_7r_25d_pml<<<n_block_front, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin, xmax, ymin, ymax, grid.z1, grid.z2,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_top(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y2-grid.y1+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            1);
        kernel_7r_25d_pml<<<n_block_top, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,grid.y1,grid.y2,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_left(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y4-grid.y3+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            1);
        kernel_7r_25d_pml<<<n_block_left, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x1,grid.x2,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_center(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y4-grid.y3+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            1);
        kernel_7r_25d_inner<<<n_block_center, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x3,grid.x4,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_right(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y4-grid.y3+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            1);
        kernel_7r_25d_pml<<<n_block_right, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            grid.x5,grid.x6,grid.y3,grid.y4,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_bottom(
            (grid.z4-grid.z3+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.y6-grid.y5+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            1);
        kernel_7r_25d_pml<<<n_block_bottom, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,grid.y5,grid.y6,grid.z3,grid.z4,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        dim3 n_block_back(
            (grid.z6-grid.z5+N_THREADS_X_DIM-1) / N_THREADS_X_DIM,
            (grid.ny+N_THREADS_Y_DIM-1) / N_THREADS_Y_DIM,
            1);
        kernel_7r_25d_pml<<<n_block_back, threadsPerBlock, 0, streams[0]>>>(
            grid.nx, grid.ny, grid.nz,
            grid.ldimx, grid.ldimy, grid.ldimz,
            xmin,xmax,ymin,ymax,grid.z5,grid.z6,
            grid.lx, grid.ly, grid.lz,
            hdx_2, hdy_2, hdz_2,
            d_u, d_v, d_vp,
            d_phi, d_eta);

        for (int i = 0; i < num_streams; i++) {
            hipStreamSynchronize(streams[i]);
        }

        kernel_add_source_kernel<<<1, 1>>>(d_v, IDX3_grid(sx,sy,sz,grid), source[istep-1]);
        clock_gettime(CLOCK_REALTIME, &end);
        *time_kernel += (end.tv_sec  - start.tv_sec) +
                        (double)(end.tv_nsec - start.tv_nsec) / 1.0e9;

        float *t = d_u;
        d_u = d_v;
        d_v = t;

        // Print out
        if (istep % npo == 0) {
            printf("time step %u / %u\n", istep, nsteps);
        }
    }


    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }


    hipMemcpy(u, d_u, gridSize(grid), hipMemcpyDeviceToHost);

    freeDeviceGrid(d_u, grid);
    freeDeviceGrid(d_v, grid);
    freeDeviceGrid(d_vp, grid);
    freeDeviceGrid(d_phi, grid);
    freeDeviceGrid(d_eta, grid);
}
